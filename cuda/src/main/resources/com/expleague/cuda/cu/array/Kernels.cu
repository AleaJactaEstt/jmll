
#include <hip/hip_runtime.h>
extern "C"
__global__ void fMatrixExp(
    const float* arguments,
    float* results,
    const int states
) {
  const int X = gridDim.x;
  const int col = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if (col < states) {
    float sum = 0;
    for (int j = 0; j < states - 1; j++) {
      results[col * states + j] = expf(arguments[col * (states - 1) + j]);
      sum = sum + results[col * states + j];
    }
    sum = sum + 1;
    results[col * states + states - 1] = 1;
    for (int j = 0; j < states; j++) {
      results[col * states + j] = results[col * states + j] / sum;
    }
  }
}

extern "C"
#define BLOCK_DIM 1024
__global__ void fMatrixReduce(
    const float* arguments,
    float* results
) {
  const int col = blockIdx.x;
  const int states = blockDim.x;
  const int tid = threadIdx.x;
  const int index = states * col + tid;
  __shared__ float sdata[BLOCK_DIM];
  __shared__ float res[BLOCK_DIM];
  if (tid < (states - 1)) {
    const float f = expf(arguments[col * (states - 1) + tid]);
    sdata[tid] = f;
    res[tid] = f;
  } else {
    sdata[tid] = 1;
    res[tid] = 1;
  }

  __syncthreads();

  for (int s = BLOCK_DIM / 2; s > 0; s>>=1) {
    if (tid < s) {
        sdata[tid] += sdata[tid + s];
    }
    __syncthreads();
  }

  results[index] = res[tid] / sdata[0];
}

extern "C"
#define BLOCK_SIZE 32
__global__ void reduce5(const float* arguments, float* results, const int n) {
    extern __shared__ float sdata[];
    const int tid = threadIdx.x;
    const int i = blockIdx.x*blockDim.x + tid;

    if (i < n) {
        sdata[tid] = arguments[i];
    }

    for (int s = BLOCK_SIZE / 2; s > 0; s>>=1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        results[blockIdx.x] = sdata[0];
    }
}


extern "C"
__global__ void fFill(
    float* arguments,
    const float value,
    const int size
) {
  const int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index < size) {
    arguments[index] = value;
  }
}

extern "C"
__global__ void fMatrixKernel1(
    const int states,
    const float weight,
    const float diff,
    const float* distribution,
    const float* expectedValue,
    float* betaGrad,
    const int to,
    const float* weights
) {
  const int index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index < states * (states - 1)) {
    const int i = index / (states - 1);
    const int j = index % (states - 1);

    const float curW = weights[i * states + to];
    const float grad = 2 * weight * diff * distribution[i] * expectedValue[to];

    if (j == to) {
      betaGrad[index] += grad * curW * (1 - curW);
    } else {
      betaGrad[index] += -grad * curW * weights[i * states + j];
    }
  }
}


extern "C"
__global__ void fMatrixKernel2(
    const int states,
    const float lambda,
    float* betaGrad,
    const int to,
    const float* weights
) {
  const int index = blockDim.x * blockIdx.x + threadIdx.x;

  if (index < (states - 1) * states) {
    const int from = index / (states - 1);
    const int j = index % (states - 1);

    const float curW = weights[from * states + to];
    const float grad = lambda * curW;

    if (j == to) {
      betaGrad[index] += grad * curW * (1 - curW);
    } else {
      betaGrad[index] += -grad * curW * weights[from * states + j];
    }
  }
}

extern "C"
__global__ void fVectorKernel1(
    const float* lastGrad,
    const float* gradCoordinate,
    const float* totalGrad,
    const float step,
    const int sumSize,
    float* result,
    const int size
) {
  const int index = blockDim.x * blockIdx.x + threadIdx.x;

  if(index < size && lastGrad[index] != 0) {
      result[index] += -step * gradCoordinate[index] * totalGrad[index] / sumSize;
  }
}

/*
extern "C"
#define STATES 6
#define SIZE 15
__global__ void getSeqValue(
    const float* params,
    const float* seq,
    const int len,
    float result
) {
  const int tid = threadIdx.x;
  const int dim = STATES * (STATES - 1) * SIZE + STATES;

  __shared__ float sdata[dim];
  __shared__ float res[BLOCK_DIM];
  if (tid < dim) {
    sdata[tid] = params[tid];
  }
  __syncthreads();

  for (int i = 0; i < len; i++) {
    const int offset = seq[i] * STATES * (STATES - 1);
    if (tid >= offset && tid < offset + STATES * (STATES - 1)) {
      const int row = (tid - offset) / STATES;

    }
    __syncthreads();
  }

  if (tid < (states - 1)) {
    for (int s = 1; s < states - 1; s *= 2) {
      if (tid % (2 * s) == 0) {
        sdata[tid] += sdata[tid + s];
      }
     __syncthreads();
    }
  }

  const float sum = sdata[0] + 1;
  results[index] = res[tid] / sum;
}*/