
#include <hip/hip_runtime.h>

__device__ float sigmoid(float x) {
    return 1 / (1 + expf(-x));
}

extern "C"
#define BLOCK_DIM 4
#define SWINDOW_DIM 8
#define WDATA_DIM 16
__global__ void produceState(const float* arguments, const int argsSize, const float* weights, 
                                const int* topology, const int topSize, float* outStates) {
    const int tid = threadIdx.x;
    const int dim = argsSize + topSize;  
    //extern __shared__ float s[];
    //float* states = s;
    //bool* ready = (bool*)&states[dim];
    //extern __shared__ bool ready[]; 
    __shared__ int counter[BLOCK_DIM];
    __shared__ float swindow[SWINDOW_DIM];
    __shared__ int tdata[BLOCK_DIM * 3];
    __shared__ float wdata[WDATA_DIM];
    int totalCount = argsSize;
    int offset = 0;

    for (int i = tid; i < argsSize; i += blockDim.x) {
        swindow[i] = arguments[i];
    }        
    counter[tid] = 0;
    __syncthreads();

    /*

    while(totalCount < dim) {
        const int t = totalCount - argsSize;
        int topCount = blockDim.x * 3;
        if (t + blockDim.x > topSize) {
            topCount = (topSize - t) * 3;
        }
        for (int i = tid; i < topCount; i += blockDim.x) {
            tdata[i] = topology[t * 3 + i]
        }

        const int topIndex = t + tid;
        if (topIndex < topSize) {
            const int leftBorder = tdata[tid];
            const int rightBorder = tdata[tid + 1];
            const int weightsStart = tdata[tid + 2];

            if (rightBorder <= counter[0]) {
                float sum = 0;
                for (int i = leftBorder; i < rightBorder; i++) {
                    sum += outStates[i] * weights[weightsStart + i - leftBorder];
                }

                outStates[index] = sigmoid(sum);
                ready[index] = true;
            }
        }
        __syncthreads();

        if (tid == 0) {
            int total = counter[0];
            for (int i = total; i < total + blockDim.x && i < dim; i++) {
                if (ready[i]) {
                    counter[0]++;
                }
            }
        } 
        __syncthreads();
    }
    */

    //int n = tid;
    //while(n < dim) {
    //    outStates[n] = states[n];
    //    n += blockDim.x;
    //}
}


extern "C"
__global__ void produceState3(const float* arguments, const int argsSize, const float* weights, 
                                const int* topology, const int topSize, float* outStates) {
    const int tid = threadIdx.x;
    const int dim = argsSize + topSize;  
    extern __shared__ float s[];
    float* states = s;
    int* iters = (int*)&states[dim];      

    iters[tid] = 0;
    int r = tid;
    while (r < argsSize) {
        states[r] = arguments[r];
        iters[tid]++;
        r += blockDim.x;
    }
    __syncthreads();

    while(iters[tid] * blockDim.x + tid < dim) {
        const int index = iters[tid] * blockDim.x + tid;
        const int topIndex = index - argsSize;
        const int leftBorder = topology[topIndex*3];
        const int rightBorder = topology[topIndex*3 + 1];
        const int weightsStart = topology[topIndex*3 + 2];

        bool canStart = true;
        for (int i = leftBorder; i < rightBorder; i++) {
            int threadId = i % blockDim.x;
            int mustCounted = i / blockDim.x + 1;
            if (iters[threadId] < mustCounted) {
                canStart = false;
                break;
            }
        }

        if (canStart) {
            float sum = 0;
            for (int i = leftBorder; i < rightBorder; i++) {
                sum += states[i] * weights[weightsStart + i - leftBorder];
            }
            states[index] = sigmoid(sum);
            iters[tid]++;
        }
        __syncthreads();
    }

    __syncthreads();

    int n = tid;
    while(n < dim) {
        outStates[n] = states[n];
        n += blockDim.x;
    }
}