
#include <hip/hip_runtime.h>
extern "C"
__global__ void dSigmoid(double *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = 1. / (1. + exp(-original[index]));
  }
}

extern "C"
__global__ void dExp(double *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = exp(original[index]);
  }
}

extern "C"
__global__ void dTanh(double *original, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = tanh(original[index]);
  }
}

extern "C"
__global__ void dRndSigmoid(double *original, double *random, int size) {
  const int X = gridDim.x;
  const int index = gridDim.y * X * threadIdx.x + X * blockIdx.y + blockIdx.x;

  if(index < size) {
    original[index] = (1. / (1. + exp(-original[index]))) > random[index];
  }
}
